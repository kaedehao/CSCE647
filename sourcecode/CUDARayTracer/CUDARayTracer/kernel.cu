#include "hip/hip_runtime.h"
// includes, system
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <iostream>
using namespace std;

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include "element.cuh"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
unsigned int window_width  = 1024;
unsigned int window_height = 768;

// vbo variables
GLuint vbo = -1;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
			   unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);

void init_scene()
{
	// initialize the scene
}

///////////////////////////////////////////////////////////////////////////////
//! main entry of the ray tracing program
///////////////////////////////////////////////////////////////////////////////
__global__ void raytrace(float3 *pos, unsigned int width, unsigned int height)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	vec3 v1((x%255) / 255.0, (y%255)/255.0, 0);
	vec3 v2(0, (x%255) / 255.0, (y%255)/255.0);
	vec3 v = v1.cross(v2);	

	Color c( v.x, v.y, v.z, 255);

	// write output vertex
	pos[y*width+x] = make_float3(x, y, c.toFloat());
}


void launch_kernel(float3 *pos, unsigned int mesh_width,
				   unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	raytrace<<< grid, block>>>(pos, window_width, window_height);
}

bool checkHW(char *name, const char *gpuType, int dev)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	strcpy(name, deviceProp.name);

	if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
	{
		return true;
	}
	else
	{
		return false;
	}
}

int findGraphicsGPU(char *name)
{
	int nGraphicsGPU = 0;
	int deviceCount = 0;
	bool bFoundGraphics = false;
	char firstGraphicsName[256], temp[256];

	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("> FAILED program finished, exiting...\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("> There are no device(s) supporting CUDA\n");
		return false;
	}
	else
	{
		printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
	}

	for (int dev = 0; dev < deviceCount; ++dev)
	{
		bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
		printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

		if (bGraphics)
		{
			if (!bFoundGraphics)
			{
				strcpy(firstGraphicsName, temp);
			}

			nGraphicsGPU++;
		}
	}

	if (nGraphicsGPU)
	{
		strcpy(name, firstGraphicsName);
	}
	else
	{
		strcpy(name, "this hardware");
	}

	return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	char *ref_file = NULL;

	pArgc = &argc;
	pArgv = argv;

	printf("%s starting...\n");

	if (argc > 1)
	{
		if (checkCmdLineFlag(argc, (const char **)argv, "file"))
		{
			// In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
			getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
		}
	}

	printf("\n");

	runTest(argc, argv, ref_file);

	hipDeviceReset();
	printf("program completed, returned %s\n", (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CUDA Ray Tracer: %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}


void showCUDAMemoryUsage() {
    size_t free_byte ;
    size_t total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
    if ( hipSuccess != cuda_status ){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
		exit(1);
    }
	
	double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

void resize(int w, int h) 
{
	if( w == window_width &&  h == window_height ) return;

	window_width = w, window_height = h;

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	showCUDAMemoryUsage();

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, window_width, 0, window_height, 0.1, 10.0);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("CUDA Ray Tracer");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);

	// initialize necessary OpenGL extensions
	glewInit();

	if (! glewIsSupported("GL_VERSION_2_0 "))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, window_width, 0, window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	{
		if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
		{
			return false;
		}
	}
	else
	{
		cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	}

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// initialize the scene on CUDA kernels
	init_scene();

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);

	glutReshapeFunc(resize);

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();
	atexit(cleanup);

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float3 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));
	//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

	// execute the kernel
	//    dim3 block(8, 8, 1);
	//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	//    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

	launch_kernel(dptr, window_width, window_height, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
	printf("sdkDumpBin: <%s>\n", filename);
	FILE *fp;
	FOPEN(fp, filename, "wb");
	fwrite(data, bytes, 1, fp);
	fflush(fp);
	fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
			   unsigned int vbo_res_flags)
{
	assert(vbo);

	if( !(*vbo) ) {
		cout << "generating new vbo..." << endl;
		// create buffer object
		glGenBuffers(1, vbo);
	}
	else {
		hipGraphicsUnregisterResource(*vbo_res);
	}
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = window_width * window_height * 3 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	// unregister this buffer object with CUDA
	hipGraphicsUnregisterResource(vbo_res);

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4,GL_UNSIGNED_BYTE,12,(GLvoid*)8);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, window_width * window_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += 0.01f;

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	glutPostRedisplay();
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27) :
		exit(EXIT_SUCCESS);
		break;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1<<button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}
