#include "hip/hip_runtime.h"
#include "Scene.h"
#include "utils.h"
#include "extras/tinyobjloader/tiny_obj_loader.h"
#include "extras/aabbtree/aabbtree.h"

Scene::Scene(void)
{
	envmap = -1;	// by default, no environment mapping
	name = "scene"; // default name
	gamma = 1.0;	// default gamma
	ttype = 1;
}


Scene::~Scene(void)
{
}


bool Scene::load(const string& filename)
{
	ifstream fin(filename);
	if( !fin ) {
		cerr << "Failed to load file " << filename << '!' << endl;
		return false;
	}
	while( fin ) {
		string line;
		cout << line << endl;
		getline(fin, line);
		parse(line);
	}
	return true;
}

bool Scene::save(const string& filename){
	return false;
}

void Scene::parse(const string& line)
{
	stringstream ss;
	ss<<line;

	string tag;
	ss >> tag;
	cout << tag << endl;

	std::for_each(tag.begin(), tag.end(), ::tolower);
	
	if( tag == "name" ) {
		ss >> name;
	}
	if( tag == "tracingtype" ) {
		ss >> ttype;
	}
	if( tag == "environment" ) {
		string texFile;
		ss >> texFile;

		// test if it is an hdr image
		if( isHDRFile(texFile) ) {
			envmap = loadHDRTexture(texFile, texs);
		}
		else {
			envmap = loadTexture(texFile.c_str(), texs);
		}
	}
	else if( tag == "maxiters" ) {
		ss >> maxiters;
	}
	else if( tag == "gamma" ) {
		ss >> gamma;
	}
	else if( tag == "canvas" ) {
		ss >> w >> h;
	}
	else if( tag == "camera" ) {
		ss >> cam.pos >> cam.dir >> cam.up >> cam.f >> cam.fov;
		cam.dir = cam.dir.normalized();
		cam.up = cam.up.normalized();
	}
	else if( tag == "material" ) {
		Material mater;
		ss >> mater.name >> mater;

		if( mater.diffuseTexName != "none" ) {
			mater.diffuseTex = TextureObject::parseType(mater.diffuseTexName);
			if( mater.diffuseTex == TextureObject::Image ){ 
				// load texture from image file
				if( mater.isSolidTex ) 
					mater.diffuseTex += loadTexture(mater.diffuseTexName.c_str(), texs);
				else
					mater.diffuseTex = loadTexture(mater.diffuseTexName.c_str(), texs);
			}
		}
		else mater.diffuseTex = -1;
		if( mater.normalTexName != "none" ) {
			mater.normalTex = TextureObject::parseType(mater.normalTexName);
			if( mater.normalTex == TextureObject::Image ){ 
				mater.normalTex = loadTexture(mater.normalTexName.c_str(), texs);
			}
		}
		else mater.normalTex = -1;

		materials.push_back(mater);
		materialMap[mater.name] = materials.size()-1;
	}
	else if( tag == "plane" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;
		
		// construct transformation matrix
		mat3 mscl = mat3::scaling(S.x, S.y, S.z);
		mat3 mrot = mat3::rotation(R.x, R.y, R.z);
		mat3 M = mscl * mrot;

		vec3 n(0, 1, 0), u(1, 0, 0), v(0, 0, 1);
		vec3 dim = mscl * vec3(1, 1, 1);

		Shape sp = Shape::createPlane(T, dim.x, dim.y, dim.z, mrot*n, mrot*u, mrot*v, materialMap[matName]);

		sp.bb.minPt = mrot * make_float3(-S.x, -1e-1, -S.z);
		sp.bb.maxPt = mrot * make_float3(S.x, 1e-1, S.z);

		shapes.push_back(sp);
	}
	else if( tag == "box" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		// construct transformation matrix
		mat3 mscl = mat3::scaling(S.x, S.y, S.z);
		mat3 mrot = mat3::rotation(R.x, R.y, R.z);
		mat3 M = mrot * mscl;

		vec3 dim = mscl * vec3(1, 1, 1);
		vec3 p, n, u, v;

		// top
		p = vec3(0, 1.0, 0), n = vec3(0, 1, 0), u = vec3(1, 0, 0), v = vec3(0, 0, 1);		
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
		// bottom
		p = vec3(0, -1.0, 0), n = vec3(0, -1, 0), u = vec3(-1, 0, 0), v = vec3(0, 0, -1);
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
		// left
		p = vec3(-1.0, 0, 0), n = vec3(-1, 0, 0), u = vec3(0, 1, 0), v = vec3(0, 0, 1);
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
		// right
		p = vec3(1.0, 0, 0), n = vec3(1, 0, 0), u = vec3(0, -1, 0), v = vec3(0, 0, -1);
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
		// front
		p = vec3(0, 0, 1.0), n = vec3(0, 0, 1), u = vec3(0, 1, 0), v = vec3(1, 0, 0);
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
		// back
		p = vec3(0, 0, -1.0), n = vec3(0, 0, -1), u = vec3(0, -1, 0), v = vec3(-1, 0, 0);
		shapes.push_back(Shape::createPlane(M*p + T, dim.x, dim.y, 1.0, mrot*n, mrot*u, mrot*v, materialMap[matName]));
	}
	else if( tag == "sphere" ) {

		vec3 T, S, R;
		string matName;
		ss >> T >> S >> R >> matName;
		cout << matName << endl;

		mat3 mscl = mat3::scaling(S.x, S.y, S.z);
		mat3 mrot = mat3::rotation(R.x, R.y, R.z);
		mat3 M = mrot * mscl;

		vec3 dim = mscl * vec3(1, 1, 1);

		Shape sp = Shape::createSphere(T, dim.x, materialMap[matName]);

		S = S * 1.5;
		sp.bb.maxPt = make_float3(T.x + S.x, T.y + S.y, T.z + S.z);
		sp.bb.minPt = make_float3(T.x - S.x, T.y - S.y, T.z - S.z);

		shapes.push_back(sp);
	}
	else if( tag == "ellipsoid") {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		mat3 mrot = mat3::rotation(R.x, R.y, R.z);


		Shape sp = Shape::createEllipsoid(T, S, mrot*vec3(1, 0, 0), mrot*vec3(0, 1, 0), mrot*vec3(0, 0, 1), materialMap[matName]);

		shapes.push_back(sp);
	}
	else if( tag == "cylinder" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		mat3 mrot = mat3::rotation(R.x, R.y, R.z);

		shapes.push_back(Shape::createCylinder(T, S, mrot*vec3(1, 0, 0), mrot*vec3(0, 1, 0), mrot*vec3(0, 0, 1), materialMap[matName]));
	}
	else if( tag == "cone" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		mat3 mrot = mat3::rotation(R.x, R.y, R.z);

		shapes.push_back(Shape::createCone(T, S, mrot*vec3(1, 0, 0), mrot*vec3(0, 1, 0), mrot*vec3(0, 0, 1), materialMap[matName]));
	}
	else if( tag == "hyperboloid" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		mat3 mrot = mat3::rotation(R.x, R.y, R.z);

		shapes.push_back(Shape::createHyperboloid(T, S, mrot*vec3(1, 0, 0), mrot*vec3(0, 1, 0), mrot*vec3(0, 0, 1), materialMap[matName]));
	}
	else if( tag == "hyperboloid2" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string matName;
		ss >> matName;

		mat3 mrot = mat3::rotation(R.x, R.y, R.z);

		shapes.push_back(Shape::createHyperboloid2(T, S, mrot*vec3(1, 0, 0), mrot*vec3(0, 1, 0), mrot*vec3(0, 0, 1), materialMap[matName]));
	}
	else if( tag == "mesh" ) {
		vec3 T, S, R;
		ss >> T >> S >> R;
		string meshFile, matName;
		ss >> meshFile >> matName;

		mat3 mscl = mat3::scaling(S.x, S.y, S.z);
		mat3 mrot = mat3::rotation(R.x, R.y, R.z);
		mat3 M = mrot * mscl;

		vec3 dim = mscl * vec3(1, 1, 1);

		Shape sp = Shape::createMesh(T, S, mrot, materialMap[matName]);
		
		// load the mesh and convert it to a texture
		vector<tinyobj::shape_t> objs;
		int lastSlashPos = meshFile.find_last_of("/");
		string basePath = meshFile.substr(0, lastSlashPos+1);
		cout << "base path: " << basePath << endl;
		cout << tinyobj::LoadObj(objs, meshFile.c_str(), basePath.c_str()) << endl;
		cout << objs.size() << " shapes in total." << endl;

		// count triangle number
		int ntris = 0;
		for(int i=0,tidx=0;i<objs.size();i++) {
			const tinyobj::shape_t& shp = objs[i];
			const tinyobj::mesh_t& msh = shp.mesh;
			ntris += msh.indices.size() / 3;
		}

		cout << "number of triangles in the mesh: " << ntris << endl;
		
		vector<float4> triangles;
		triangles.reserve(ntris);
		vector<aabbtree::Triangle> tris;		// for building AABB tree
		tris.reserve(ntris);
		vector<float3> normals;
		normals.reserve(ntris);
		vector<float2> texcoords;
		texcoords.reserve(ntris);
		cout << "space reserved for processing the mesh." << endl;

		float3 maxPt = make_float3(-FLT_MAX), minPt = make_float3(FLT_MAX);

		for(int i=0,tidx=0;i<objs.size();i++) {
			const tinyobj::shape_t& shp = objs[i];

			const tinyobj::mesh_t& msh = shp.mesh;
			const tinyobj::material_t& mt = shp.material;

			bool hasNormal = !msh.normals.empty();
			bool hasTexCoords = !msh.texcoords.empty();

			for(int j=0;j<msh.indices.size();j+=3) {
				float3 v0 = make_float3(msh.positions[msh.indices[j]*3], msh.positions[msh.indices[j]*3+1], msh.positions[msh.indices[j]*3+2]);
				float3 v1 = make_float3(msh.positions[msh.indices[j+1]*3], msh.positions[msh.indices[j+1]*3+1], msh.positions[msh.indices[j+1]*3+2]);
				float3 v2 = make_float3(msh.positions[msh.indices[j+2]*3], msh.positions[msh.indices[j+2]*3+1], msh.positions[msh.indices[j+2]*3+2]);

				v0 = M * v0 + T.data;
				v1 = M * v1 + T.data;
				v2 = M * v2 + T.data;

				maxPt = fmaxf(v0, maxPt); minPt = fminf(v0, minPt);
				maxPt = fmaxf(v1, maxPt); minPt = fminf(v1, minPt);
				maxPt = fmaxf(v2, maxPt); minPt = fminf(v2, minPt);
				
				triangles.push_back(make_float4(v0, i));
				triangles.push_back(make_float4(v1, i));
				triangles.push_back(make_float4(v2, i));

				float3 n0 = aabbtree::zero3, n1 = aabbtree::zero3, n2 = aabbtree::zero3;
				if( hasNormal ) {
					n0 = make_float3(msh.normals[msh.indices[j]*3], msh.normals[msh.indices[j]*3+1], msh.normals[msh.indices[j]*3+2]);
					n1 = make_float3(msh.normals[msh.indices[j+1]*3], msh.normals[msh.indices[j+1]*3+1], msh.normals[msh.indices[j+1]*3+2]);
					n2 = make_float3(msh.normals[msh.indices[j+2]*3], msh.normals[msh.indices[j+2]*3+1], msh.normals[msh.indices[j+2]*3+2]);

					n0 = normalize(mrot * n0);
					n1 = normalize(mrot * n1);
					n2 = normalize(mrot * n2);

					normals.push_back(n0);
					normals.push_back(n1);
					normals.push_back(n2);
				}
				//tris.push_back(aabbtree::Triangle(tidx++, v0, v1, v2, n0, n1, n2));
				tris.push_back(aabbtree::Triangle(tidx++, v0, v1, v2));
			}
		}

		sp.bb.minPt = minPt;
		sp.bb.maxPt = maxPt;

		cout << "uploading vertices ..." << endl;
		cout << "copying " << bytes2MB(sizeof(float4)*triangles.size()) << " MB to GPU ..." << endl;
		hipMalloc(&sp.trimesh.faces, sizeof(float4)*triangles.size());
		hipMemcpy(sp.trimesh.faces, &triangles[0], sizeof(float4)*triangles.size(), hipMemcpyHostToDevice);

		if( normals.empty() )
			sp.trimesh.normals = NULL;
		else {
			cout << "uploading normals ..." << endl;
			cout << "copying " << bytes2MB(sizeof(float3)*normals.size()) << " MB to GPU ..." << endl;

			hipMalloc(&sp.trimesh.normals, sizeof(float3)*normals.size());
			hipMemcpy(sp.trimesh.normals, &normals[0], sizeof(float3)*normals.size(), hipMemcpyHostToDevice);
		}
		if( texcoords.empty() ) 
			sp.trimesh.texcoords = NULL;
		else {
			cout << "uploading texture coordinates ..." << endl;
			cout << "copying " << bytes2MB(sizeof(float2)*texcoords.size()) << " MB to GPU ..." << endl;

			hipMalloc(&sp.trimesh.texcoords, sizeof(float2)*texcoords.size());
			hipMemcpy(sp.trimesh.texcoords, &texcoords[0], sizeof(float2)*texcoords.size(), hipMemcpyHostToDevice);
		}

		sp.trimesh.nFaces = triangles.size()/3;

		cout << sp.trimesh.faces << ' '
			 << sp.trimesh.normals << ' '
			 << sp.trimesh.texcoords << ' '
			 << sp.trimesh.nFaces << endl;
		
		// release some memory
		triangles.clear();
		normals.clear();
		texcoords.clear();

		aabbtree::AABBTree tree(tris);
		tree.printNodeStats();
		auto treearray = tree.toArray();

		cout << "uploading aabb tree to device ..." << endl;
		// upload the tree to device
		size_t treesize = sizeof(aabbtree::AABBNode_Serial)*treearray.size();
		cout << "tree size = " << bytes2MB(treesize) << " MB" << endl;
		hipMalloc(&sp.trimesh.tree, treesize);		
		hipMemcpy(sp.trimesh.tree, &treearray[0], treesize, hipMemcpyHostToDevice);
		cout << "done." << endl;

		shapes.push_back(sp);
	}
	else return;
}
