#include "hip/hip_runtime.h"
// includes, system
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <ctime>
#include <iostream>
using namespace std;

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include <hip/hip_texture_types.h>

#include <hip/hip_vector_types.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "element.h"
#include "definitions.h"
#include "Scene.h"
#include "trackball.h"

#include "FreeImagePlus.h"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD         0.30f
#define REFRESH_DELAY     25 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
unsigned int edgeX = 8, edgeY = 8;
unsigned int window_width  = 1024 + edgeX;
unsigned int window_height = 768 + edgeY;

// vbo variables
GLuint vbo = 0;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;
TrackBall tball;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
			   unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
extern __global__ void setParams(int, int, int);
extern __global__ void bindTexture2(const hipTextureObject_t* texs, int texCount);

extern __global__ void copy2pbo(float3*, float3*, int, int, int, float);
extern __global__ void clearCumulatedColor(float3*, int, int);

extern __global__ void raytrace(float time, float3 *pos, Camera* cam, 
						 int nLights, int* lights, 
						 int nShapes, Shape* shapes,
						 int nMaterials, Material* materials,
						 unsigned int width, unsigned int height,
						 int sMode, int AASamples);

extern __global__ void raytrace2(float time, float3 *pos, Camera* cam, 
						 int nLights, int* lights, 
						 int nShapes, Shape* shapes, 
						 int nMaterials, Material* materials,
						 unsigned int width, unsigned int height,
						 int sMode, int AASamples, 
						 int gx, int gy, int gmx, int gmy);

extern __global__ void initCurrentBlock(int v);

extern __global__ void raytrace3(float time, float3 *pos, Camera* cam, 
						 int nLights, int* lights, 
						 int nShapes, Shape* shapes, 
						 int nMaterials, Material* materials,
						 unsigned int width, unsigned int height,
						 int sMode, int AASamples, 
						 int bmx, int bmy, int tlb);

void runCuda(struct hipGraphicsResource **vbo_resource);


Scene scene;

Camera cam;
Camera* d_cam;
vector<Shape> shapes;
Shape* d_shapes;
TextureObject* d_tex;
hipTextureObject_t* d_texobjs;
vector<int> lights;
int* d_lights;
vector<Material> materials;
Material* d_materials;
float3* cumulatedColor = 0;
int AASamples = 1;
int sMode = 1;
int kernelIdx = 0;
int specType = 0;
int tracingType = 0;
int iterations = 0;
float gamma = 1.0;

void init_scene()
{
	showCUDAMemoryUsage();
	const size_t sz = sizeof(Camera);
	hipMalloc((void**)&d_cam, sz);
	hipMemcpy(d_cam, &cam, sz, hipMemcpyHostToDevice);

	if(!scene.load("scene0.txt")) cout << "scene file loading failed!" << endl;
	else {
		shapes = scene.getShapes();
		materials = scene.getMaterials();
		lights = scene.getLights();
		cout << "scene loaded. " 
			 << shapes.size() << " shapes in total." 
			 << materials.size() << " materials in total." 
			 << scene.getTextures().size() << " textures in total." 
			 << endl;
	}

	cam = scene.camera();

	hipMalloc((void**)&d_cam, sizeof(Camera));
	hipMemcpy(d_cam, &cam, sizeof(Camera), hipMemcpyHostToDevice);

	const size_t sz_shapes = shapes.size() * sizeof(Shape);
	hipMalloc((void**)&d_shapes, sz_shapes);
	hipMemcpy(d_shapes, &(shapes[0]), sz_shapes, hipMemcpyHostToDevice);

	const size_t sz_mats = materials.size() * sizeof(Material);
	hipMalloc((void**)&d_materials, sz_mats);
	hipMemcpy(d_materials, &(materials[0]), sz_mats, hipMemcpyHostToDevice);

	const size_t sz_lights = lights.size() * sizeof(int);
	hipMalloc((void**)&d_lights, sz_lights);
	hipMemcpy(d_lights, &(lights[0]), sz_lights, hipMemcpyHostToDevice);

	const vector<TextureObject>& texs = scene.getTextures();
	const size_t sz_tex = sizeof(TextureObject)*texs.size();
	cout << "sz_tex = " << sz_tex << endl;
	hipMalloc((void**)&d_tex, sz_tex);
	hipMemcpy(d_tex, &(texs[0]), sz_tex, hipMemcpyHostToDevice);

	// create texture objects for textures
	vector<hipTextureObject_t> texobjs;
	for(int i=0;i<texs.size();i++) {
		// Allocate CUDA array in device memory
		hipChannelFormatDesc channelDesc;
		if( texs[i].isHDR )
			channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
		else
			channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
		hipArray* cuArray;
		hipMallocArray(&cuArray, &channelDesc, texs[i].size.x, texs[i].size.y);

		// Copy to device memory some data located at address h_data
		// in host memory
		if( texs[i].isHDR )
			hipMemcpyToArray(cuArray, 0, 0, texs[i].addr, texs[i].size.x*texs[i].size.y*sizeof(float4), hipMemcpyDeviceToDevice);
		else
			hipMemcpyToArray(cuArray, 0, 0, texs[i].addr, texs[i].size.x*texs[i].size.y*sizeof(uchar4), hipMemcpyDeviceToDevice);

		// create texture object
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		// Specify texture object parameters
		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.addressMode[0]   = hipAddressModeWrap;
		texDesc.addressMode[1]   = hipAddressModeWrap;
		texDesc.filterMode       = hipFilterModeLinear;
		if( texs[i].isHDR )
			texDesc.readMode         = hipReadModeElementType;
		else
			texDesc.readMode         = hipReadModeNormalizedFloat;
		texDesc.normalizedCoords = 1;

		// create texture object: we only have to do this once!
		hipTextureObject_t tex=0;
		hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

		texobjs.push_back(tex);
	}
	size_t sz_texobjs = sizeof(hipTextureObject_t)*texobjs.size();
	hipMalloc((void**)&d_texobjs, sz_texobjs);
	hipMemcpy(d_texobjs, &(texobjs[0]), sz_texobjs, hipMemcpyHostToDevice);

	cout << "scene initialized." << endl;
}

void launch_kernel(float3 *pos, unsigned int mesh_width,
				   unsigned int mesh_height, int sMode)
{
	// update camera info	
	mat4 mat(tball.getInverseMatrix());
	mat = mat.trans();

	vec3 camPos = cam.pos;
	vec3 camDir = cam.dir;
	vec3 camUp = cam.up;

	camPos = (mat * (camPos / tball.getScale()));
	camDir = (mat * camDir);
	camUp = (mat * camUp);

	Camera caminfo = cam;
	caminfo.dir = camDir;
	caminfo.up = camUp;
	caminfo.pos = camPos;
	caminfo.right = caminfo.dir.cross(caminfo.up);
	
	hipMemcpyAsync(d_cam, &caminfo, sizeof(Camera), hipMemcpyHostToDevice);

	bindTexture2<<< 1, 1 >>>(d_texobjs, scene.getTextures().size());
	setParams<<<1, 1>>>(specType, tracingType, scene.getEnvironmentMap());

	switch( kernelIdx ) {
	case 0:{
		// execute the kernel
		dim3 block(32, 32, 1);
		dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
		raytrace<<< grid, block >>>((iterations+rand()%1024), cumulatedColor, d_cam,
			lights.size(), d_lights,
			shapes.size(), d_shapes,
			materials.size(), d_materials,
			window_width, window_height, sMode, AASamples);
		break;
		   }
	case 1:{
		dim3 block(32, 32, 1);
		dim3 group(4, 4, 1);
		dim3 grid(group.x, group.y, 1);
		dim3 groupCount(ceil(window_width/(float)(block.x * group.x)), ceil(window_height/(float)(block.y * group.y)), 1);

		raytrace2<<< grid, block >>>((iterations+rand()%1024), cumulatedColor, d_cam,
			lights.size(), d_lights,
			shapes.size(), d_shapes,
			materials.size(), d_materials,
			window_width, window_height, sMode, AASamples, 
			group.x, group.y, groupCount.x, groupCount.y);
		break;
		   }
	case 2:{
		dim3 block(32, 32, 1);
		dim3 grid(4, 4, 1);

		dim3 blockCount(ceil(window_width/(float)block.x), ceil(window_height/(float)block.y ), 1);

		unsigned totalBlocks = blockCount.x*blockCount.y;
		//cout << "total blocks = " << totalBlocks << endl;
		srand(clock());

		initCurrentBlock<<<1, 1>>>(0);
		raytrace3<<< grid, block >>>((iterations+rand()%1024), cumulatedColor, d_cam,
			lights.size(), d_lights,
			shapes.size(), d_shapes,
			materials.size(), d_materials,
			window_width, window_height, sMode, AASamples, 
			blockCount.x, blockCount.y, totalBlocks);
		break;
		   }
	}
	hipDeviceSynchronize();

	iterations++;
	//cout << iterations << endl;

	// copy to pbo
	dim3 block(32, 32, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	copy2pbo<<<grid,block>>>(cumulatedColor, pos, iterations, window_width, window_height, gamma);
	hipDeviceSynchronize();
}

bool checkHW(char *name, const char *gpuType, int dev)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	strcpy(name, deviceProp.name);

	if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
	{
		return true;
	}
	else
	{
		return false;
	}
}

int findGraphicsGPU(char *name)
{
	int nGraphicsGPU = 0;
	int deviceCount = 0;
	bool bFoundGraphics = false;
	char firstGraphicsName[256], temp[256];

	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("> FAILED program finished, exiting...\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("> There are no device(s) supporting CUDA\n");
		return false;
	}
	else
	{
		printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
	}

	for (int dev = 0; dev < deviceCount; ++dev)
	{
		bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
		printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

		if (bGraphics)
		{
			if (!bFoundGraphics)
			{
				strcpy(firstGraphicsName, temp);
			}

			nGraphicsGPU++;
		}
	}

	if (nGraphicsGPU)
	{
		strcpy(name, firstGraphicsName);
	}
	else
	{
		strcpy(name, "this hardware");
	}

	return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	FreeImage_Initialise();

	srand(clock());
	
	char *ref_file = NULL;

	pArgc = &argc;
	pArgv = argv;

	//printf("%s starting...\n");

	if (argc > 1)
	{
		if (checkCmdLineFlag(argc, (const char **)argv, "file"))
		{
			// In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
			getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
		}
	}

	printf("\n");

	runTest(argc, argv, ref_file);

	FreeImage_DeInitialise();
	return 0;
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CUDA Ray Tracer: %3.1f fps - Iteration %d", avgFPS, iterations);
	glutSetWindowTitle(fps);
}


void resize(int w, int h) 
{
	tball.reshape(w, h);
	//return;
	cout << w << "x" << h << " vs " << window_width << "x" << window_height << endl;

	if( w == window_width &&  h == window_height ) return;

	window_width = w, window_height = h;
	// camera
	cam.h = h / (float) w;

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	showCUDAMemoryUsage();

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, window_width, 0, window_height, 0.1, 10.0);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_SINGLE );
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("CUDA Ray Tracer");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	//glutTimerFunc(REFRESH_DELAY, timerEvent,0);

	// initialize necessary OpenGL extensions
	glewInit();

	if (! glewIsSupported("GL_VERSION_2_0 "))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, window_width, 0, window_height, 0.1, 10.0);

	tball.init();
	tball.setSceneScale(1.0);

	SDK_CHECK_ERROR_GL();

	return true;
}

void refresh() {
	//system("pause");
	glutPostRedisplay();
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	{
		if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
		{
			return false;
		}
	}
	else
	{
		cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	}

	
	//size_t nStack;
	//hipDeviceGetLimit(&nStack, hipLimitStackSize);
	//cout << "stack size = " << nStack << endl;
	//hipDeviceSetLimit(hipLimitStackSize, 65536);
	//hipDeviceGetLimit(&nStack, hipLimitStackSize);
	//cout << "stack size = " << nStack << endl;
	

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// initialize the scene on CUDA kernels
	init_scene();
	
	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);	

	glutReshapeFunc(resize);
	glutIdleFunc(refresh);

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();
	atexit(cleanup);

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float3 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));
	//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

	launch_kernel(dptr, window_width, window_height, sMode);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
	printf("sdkDumpBin: <%s>\n", filename);
	FILE *fp;
	FOPEN(fp, filename, "wb");
	fwrite(data, bytes, 1, fp);
	fflush(fp);
	fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
			   unsigned int vbo_res_flags)
{
	assert(vbo);

	if( !(*vbo) ) {
		cout << "generating new vbo..." << endl;
		// create buffer object
		glGenBuffers(1, vbo);
	}
	else {
		cout << "unregister vbo ..." << endl;
		hipGraphicsUnregisterResource(*vbo_res);
	}
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = window_width * window_height * 3 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	SDK_CHECK_ERROR_GL();

	// allocate memory
	if( cumulatedColor ) hipFree(cumulatedColor);
	int sz = window_width * window_height * sizeof(float3);
	hipMalloc((void**)&cumulatedColor, sz);
	hipMemset(cumulatedColor, 0, sz);
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	// unregister this buffer object with CUDA
	hipGraphicsUnregisterResource(vbo_res);

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, -1.0);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4,GL_UNSIGNED_BYTE,12,(GLvoid*)8);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, window_width * window_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	//glutSwapBuffers();
	glFlush();

	sdkStopTimer(&timer);
	computeFPS();
}

void clearColor() {
	dim3 block(32, 32, 1);
	dim3 grid(window_width / block.x, window_height / block.y, 1);
	clearCumulatedColor<<<grid,block>>>(cumulatedColor, window_width, window_height);
	iterations = 0;
	hipDeviceSynchronize();
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}

	hipFree(d_cam);
	hipFree(d_shapes);
	hipFree(d_lights);

	hipDeviceReset();
	printf("program completed, returned %s\n", (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void screenshot() {
	// Make the BYTE array, factor of 3 because it's RBG.
	int width = window_width - edgeX;
	int height = window_height - edgeX;
	BYTE* pixels = new BYTE[ 3 * width * height];
	glReadPixels(0, 0, width, height, GL_RGB, GL_UNSIGNED_BYTE, pixels);

	cv::Mat m( height, width, CV_8UC3, pixels );
	cv::cvtColor(m, m, CV_RGB2BGR);
	cv::flip(m, m, 0);
	cv::imwrite("screenshot.png", m);
	delete[] pixels;
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case '1':
	case '2':
	case '3':
		sMode = key - '0';
		glutPostRedisplay();
		break;
	case 'g':
	case 'G':
		cout << "input gamma value: " << endl;
		cin >> gamma;
		glutPostRedisplay();
		break;
	case 'a':
	case 'A':
		cout << "Please input number of samples:" << endl;
		cin >> AASamples;
		glutPostRedisplay();
		break;
	case 'k':
	case 'K':
		kernelIdx = (kernelIdx + 1) % 3;
		cout << "using kernel #" << kernelIdx << endl;
		glutPostRedisplay();
		break;
	case 't':
	case 'T':
		tracingType = (tracingType + 1) % 3;
		cout << "tracing type = " << tracingType << endl;
		clearColor();
		glutPostRedisplay();
		break;
	case 's':
	case 'S':
		specType = (specType + 1) % 5;
		glutPostRedisplay();
		break;
	case 'c':
	case 'C':
		screenshot();
		break;
	case (27) :
		cleanup();
		glutLeaveMainLoop();
		break;
	}
}

int AASamples_old;
////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1<<button;
		AASamples_old = AASamples;
		AASamples = 1;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
		AASamples = AASamples_old;
	}

	if (mouse_buttons & 1)
	{
		tball.mouse_rotate(x, y);
	}
	else if (mouse_buttons & 4)
	{		
	}

	clearColor();

	mouse_old_x = x;
	mouse_old_y = y;
	glutPostRedisplay();
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		tball.motion_rotate(x, y);
	}
	else if (mouse_buttons & 4)
	{
		tball.wheel( y - mouse_old_y );
	}

	mouse_old_x = x;
	mouse_old_y = y;

	clearColor();
	glutPostRedisplay();
}
